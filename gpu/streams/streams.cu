#include "hip/hip_runtime.h"
#include "common.h"

__global__ void
vectorAdd(int *A, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N) {
		A[i] *= A[i];
	}
}

void gpu_run(int *h_A)
{
	hipStream_t stream_1;
		
	int *d_A = NULL;
	int err = hipSuccess;

	// Per stage measurements
	long long start_cycles_h2d = 0, start_cycles_kernel = 0, start_cycles_d2h = 0;
	long long end_cycles_h2d = 0, end_cycles_kernel = 0, end_cycles_d2h = 0;
	long long tot_cycles_h2d = 0, tot_cycles_kernel = 0, tot_cycles_d2h = 0;

	// Full execution measurements
	long long start_cycles = 0, end_cycles = 0, tot_cycles = 0;

	int i = 0;

	int threadsPerBlock = 256;
	int blocksPerGrid = (NUM_PKTS + threadsPerBlock - 1) / threadsPerBlock;

	err = hipStreamCreate(&stream_1);
	CPE(err != hipSuccess, "Failed to create cudaStream\n", -1);

	err = hipMalloc((void **) &d_A, NUM_PKTS * sizeof(int));
	CPE(err != hipSuccess, "Failed to hipMalloc\n", -1);

	// Measure host-to-device memcpy latency
	for(i = 0; i < ITERS; i ++) {
		start_cycles = get_cycles();
		
		// Stage 1: host to device memcpy
		start_cycles_h2d = get_cycles();
		hipMemcpyAsync(d_A, h_A, NUM_PKTS * sizeof(int), hipMemcpyHostToDevice, stream_1);
		end_cycles_h2d = get_cycles();

		// Stage 2: kernel execution
		start_cycles_kernel = get_cycles();
		vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream_1>>>(d_A, NUM_PKTS);
		end_cycles_kernel = get_cycles();

		// Stage 2: device to host memcpy
		start_cycles_d2h = get_cycles();
		hipMemcpyAsync(h_A, d_A, NUM_PKTS * sizeof(int), hipMemcpyDeviceToHost, stream_1);
		end_cycles_d2h = get_cycles();

		// Complete full execution: the time for this is not included in per-stage measurement
		hipStreamSynchronize(stream_1);

		end_cycles = get_cycles();

		tot_cycles_h2d += (end_cycles_h2d - start_cycles_h2d);
		tot_cycles_kernel += (end_cycles_kernel - start_cycles_kernel);
		tot_cycles_d2h += (end_cycles_d2h - start_cycles_d2h);
		tot_cycles += (end_cycles - start_cycles);

		if(rand() % 100 == 0) {
			printf("%d | h2d = %d ns, kernel = %d ns, d2h = %d ns, full = %d ns\n", i, 
				(int) ((end_cycles_h2d - start_cycles_h2d) / 2.7),
				(int) ((end_cycles_kernel - start_cycles_kernel) / 2.7),
				(int) ((end_cycles_d2h - start_cycles_d2h) / 2.7),
				(int) ((end_cycles - start_cycles) / 2.7));
		}
		
		err = hipGetLastError();
		CPE(err != hipSuccess, "Fail!\n", -1);
	}

	hipFree(d_A);

	printf("\nFull execution stats:\n");
	printf("\th2d = %d ns, kernel = %d ns, d2h = %d ns, full execution = %d ns\n", 
		(int) (tot_cycles_h2d / (2.7 * ITERS)),
		(int) (tot_cycles_kernel / (2.7 * ITERS)),
		(int) (tot_cycles_d2h / (2.7 * ITERS)), 
		(int) (tot_cycles / (2.7 * ITERS)));

	long long total_busy_cycles = tot_cycles_h2d + tot_cycles_kernel + tot_cycles_d2h;
	printf("\nSynchronization time = %d ns\n", 
		(int) ((tot_cycles - total_busy_cycles) / (2.7 * ITERS)));

}

int main(void)
{
	int *h_A;
	int err = hipSuccess;

	printDeviceProperties();

	// Allocate host vectors
	err = hipHostMalloc((void **) &h_A, NUM_PKTS * sizeof(int));
	CPE(err != hipSuccess, "Could not allocate pinned memory\n", -1);

	// Verify that allocations succeeded
	if (h_A == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors
	for (int i = 0; i < NUM_PKTS; ++i)	{
		h_A[i] = i;
	}

	gpu_run(h_A);
	
	// Free pinned host memory
	hipHostFree(h_A);

	// Reset the device and exit
	err = hipDeviceReset();
	CPE(err != hipSuccess, "Failed to de-initialize the device\n", -1);

	printf("Done\n");
	return 0;
}


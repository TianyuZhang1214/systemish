#include "hip/hip_runtime.h"
#include "common.h"

__global__ void
vectorAdd(int *A, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N) {
		A[i] *= A[i];
	}
}

void gpu_run(int *h_A)
{
	hipStream_t stream_1;
		
	int *d_A = NULL;
	int err = hipSuccess;
	long long startCycles = 0, endCycles = 0, totCycles = 0;
	int i = 0;

	int threadsPerBlock = 256;
	int blocksPerGrid = (NUM_PKTS + threadsPerBlock - 1) / threadsPerBlock;

	err = hipStreamCreate(&stream_1);
	CPE(err != hipSuccess, "Failed to create cudaStream\n", -1);

	err = hipMalloc((void **) &d_A, NUM_PKTS * sizeof(int));
	CPE(err != hipSuccess, "Failed to hipMalloc\n", -1);

	// Measure host-to-device memcpy latency
	for(i = 0; i < ITERS; i ++) {
		startCycles = get_cycles();

		hipMemcpyAsync(d_A, h_A, NUM_PKTS * sizeof(int), hipMemcpyHostToDevice, stream_1);

		vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream_1>>>(d_A, NUM_PKTS);

		hipMemcpyAsync(h_A, d_A, NUM_PKTS * sizeof(int), hipMemcpyDeviceToHost, stream_1);

		hipStreamSynchronize(stream_1);

		endCycles = get_cycles();
		totCycles += (endCycles - startCycles);

		if(rand() % 100 == 0) {
			printf("%d: Single execution time: %f ns\n", i, (endCycles - startCycles) / 2.7);
		}
		
		err = hipGetLastError();
		CPE(err != hipSuccess, "Fail!\n", -1);
	}


	hipFree(d_A);

	printf("Full execution stats:\n");
	printf("\tcycles = %lld, nanoseconds = %f ns\n\n", totCycles / ITERS,
		totCycles / (ITERS * 2.7));

}

int main(void)
{
	int *h_A;
	int err = hipSuccess;

	printDeviceProperties();

	// Allocate host vectors
	err = hipHostMalloc((void **) &h_A, NUM_PKTS * sizeof(int));
	CPE(err != hipSuccess, "Could not allocate pinned memory\n", -1);

	// Verify that allocations succeeded
	if (h_A == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors
	for (int i = 0; i < NUM_PKTS; ++i)	{
		h_A[i] = i;
	}

	gpu_run(h_A);
	
	// Free pinned host memory
	hipHostFree(h_A);

	// Reset the device and exit
	err = hipDeviceReset();
	CPE(err != hipSuccess, "Failed to de-initialize the device\n", -1);

	printf("Done\n");
	return 0;
}


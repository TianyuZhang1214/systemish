#include "hip/hip_runtime.h"
#include "common.h"

int volatile *h_A, *h_B;
int volatile *d_A, *d_B;

pthread_t thread;

__global__ void
vectorAdd(volatile int *A, volatile int *B, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i < N) {
		while(1) {
			while(A[i] == 0) {
				// do nothing
			}
			if(A[i] == 3185) {
				B[i] = A[i] * A[i];
				A[i] = 0;
				break;
			}
			B[i] = A[i] * A[i];
			A[i] = 0;
		}
	}
}

void *gpu_run(void *ptr)
{
	// Per stage measurements
	long long start_cycles_h2d = 0, start_cycles_kernel = 0, start_cycles_d2h = 0;
	long long end_cycles_h2d = 0, end_cycles_kernel = 0, end_cycles_d2h = 0;
	long long tot_cycles_h2d = 0, tot_cycles_kernel = 0, tot_cycles_d2h = 0;

	// Full execution measurements
	long long start_cycles = 0, end_cycles = 0, tot_cycles = 0;

	int i = 0, j = 0;

	assert(NUM_PKTS < 64);			// Use one block

	for(i = 0; i < ITERS; i ++) {
		usleep(200000);
		start_cycles = get_cycles();
		
		// Stage 1: host to device latency
		printf("Making h_A non-zero\n");
		start_cycles_h2d = get_cycles();
		for(j = 0; j < NUM_PKTS; j ++) {
			h_A[j] = (i & 0xff) + j + 1;		// Always > 0

			if(i == ITERS - 1) {
				printf("Last iter: using 3185\n");
				h_A[j] = 3185;
			}

			assert(h_A[j] != 0);
		}
		end_cycles_h2d = get_cycles();

		while(true) {
			int all_zero = true;
			for(j = 0; j < NUM_PKTS; j ++) {
				if(h_A[j] != 0) {
					all_zero = false;
					printf("Iter %d: waiting for element %d to become 0\n", i, j);
					usleep(200000);
				}
			}

			// Check the output if the GPU kernel has processed all h_A
			if(all_zero) {
				break;
			}
		}

		// Stage 2: device to host latency
		start_cycles_d2h = get_cycles();
		for(j = 0; j < NUM_PKTS; j ++) {
			int kernel_inp = (i & 0xff) + j + 1;
			if(i != (ITERS - 1) && h_B[j] != kernel_inp * kernel_inp) {
				fprintf(stderr, "Kernel output mismatch error\n");
				exit(-1);
			}
		}
		end_cycles_d2h = get_cycles();

		end_cycles = get_cycles();

		tot_cycles_h2d += (end_cycles_h2d - start_cycles_h2d);
		tot_cycles_kernel += (end_cycles_kernel - start_cycles_kernel);
		tot_cycles_d2h += (end_cycles_d2h - start_cycles_d2h);
		tot_cycles += (end_cycles - start_cycles);

		if(rand() % 100 == 0) {
			printf("Iter %d | "
				"h2d = %d ns, kernel = %d ns, d2h = %d ns, full = %d ns\n",
				i, 
				(int) ((end_cycles_h2d - start_cycles_h2d) / 2.7),
				(int) ((end_cycles_kernel - start_cycles_kernel) / 2.7),
				(int) ((end_cycles_d2h - start_cycles_d2h) / 2.7),
				(int) ((end_cycles - start_cycles) / 2.7));
		}
	}

	printf("\nFull execution stats:\n");
	printf("\th2d = %d ns, kernel = %d ns, d2h = %d ns, full execution = %d ns\n",
		(int) (tot_cycles_h2d / (2.7 * ITERS)),
		(int) (tot_cycles_kernel / (2.7 * ITERS)),
		(int) (tot_cycles_d2h / (2.7 * ITERS)), 
		(int) (tot_cycles / (2.7 * ITERS)));

	long long total_busy_cycles = tot_cycles_h2d + tot_cycles_kernel + tot_cycles_d2h;
	printf("\nSynchronization time = %d ns\n", 
		(int) ((tot_cycles - total_busy_cycles) / (2.7 * ITERS)));

	return 0;
}

int main(void)
{
	int err = hipSuccess;
	printDeviceProperties();

	hipSetDeviceFlags(hipDeviceMapHost);

	// Allocate host vectors as mapped memory
	err = hipHostAlloc(&h_A, NUM_PKTS * sizeof(int), hipHostMallocMapped);
	err = hipHostAlloc(&h_B, NUM_PKTS * sizeof(int), hipHostMallocMapped);
	CPE(err != hipSuccess, "Could not allocate managed memory\n", -1);

	assert(h_A != NULL);
	assert(h_B != NULL);

	// Zero out the mapped memory vectors
	for(int j = 0; j < NUM_PKTS; j++)	{
		h_A[j] = 0;
		h_B[j] = 0;
	}

	// Get device pointer for mapped memory
	err = hipHostGetDevicePointer((void **) &d_A, (void *) h_A, 0);
	err = hipHostGetDevicePointer((void **) &d_B, (void *) h_B, 0);

	CPE(err != hipSuccess, "Could not get device pointer for mapped memory\n", -1);

	// Launch the CPU code
	pthread_create(&thread, NULL, gpu_run, NULL);

	// Launch the kernel once
	printf("Launching CUDA kernel\n");
	int threadsPerBlock = NUM_PKTS;
	int blocksPerGrid = (NUM_PKTS + threadsPerBlock - 1) / threadsPerBlock;
	hipStream_t my_stream;
	err = hipStreamCreate(&my_stream);
	CPE(err != hipSuccess, "Failed to create cudaStream\n", -1);

	vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, my_stream>>>(d_A, d_B, NUM_PKTS);
	hipStreamQuery(my_stream);

	printf("Waiting for CPU thread to finish\n");
	pthread_join(thread, NULL);

	

	// Free allocated mapped memory
	hipHostFree((void *) h_A);
	hipHostFree((void *) h_B);

	// Reset the device and exit
	err = hipDeviceReset();
	CPE(err != hipSuccess, "Failed to de-initialize the device\n", -1);

	printf("Done\n");
	return 0;
}


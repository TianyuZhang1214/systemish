#include "hip/hip_runtime.h"
#include "common.h"

int volatile *h_A, *h_B;
int volatile *d_A, *d_B;
int volatile *h_flag_1, *d_flag_1;
int volatile *h_flag_2, *d_flag_2;

pthread_t thread;

__global__ void
vectorAdd(volatile int *A, volatile int *B, volatile int *flag_1, volatile int *flag_2, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int iter = 0;
	
	if(i < N) {
		// Wait for CPU flag a finite number of times
		for(iter = 0; iter < ITERS; iter ++) {
			while(flag_1[0] == 0) {
				// Wait for host flag to be raised
			}

			// Lower host flag
			flag_1[0] = 0;

			B[i] = A[i] * A[i];

			// Raise device flag
			flag_2[0] = 1;
		}
	}
}

void *gpu_run(void *ptr)
{
	// Full execution measurements
	long long start_cycles = 0, end_cycles = 0, tot_cycles = 0;

	int i = 0, j = 0;

	assert(NUM_PKTS < 64);			// Use one block

	for(i = 0; i < ITERS; i ++) {
		printf("Iteration %d\n", i);

		start_cycles = get_cycles();
		
		for(j = 0; j < NUM_PKTS; j ++) {
			h_A[j] = (i & 0xff) + j + 1;		// Always > 0
			assert(h_A[j] != 0);
		}

		// Raise host flag
		h_flag_1[0] = 1;

		while(h_flag_2[0] == 0) {
			// Wait for device flag
		}

		// Lower device flag
		h_flag_2[0] = 0;

		for(j = 0; j < NUM_PKTS; j ++) {
			int kernel_inp = (i & 0xff) + j + 1;
			// Verify for all iterations except the last iteration
			if(h_B[j] != kernel_inp * kernel_inp) {
				fprintf(stderr, "Kernel output mismatch error\n");
				exit(-1);
			}
		}

		end_cycles = get_cycles();

		tot_cycles += (end_cycles - start_cycles);

		if(rand() % 100 == 0) {
			printf("Iter %d: %d ns\n", i, (int) ((end_cycles - start_cycles) / 2.7));
		}
	}

	printf("\nFull execution stats: %d ns\n", (int) (tot_cycles / (2.7 * ITERS)));

	return 0;
}

int main(void)
{
	int err = hipSuccess;
	printDeviceProperties();

	hipSetDeviceFlags(hipDeviceMapHost);

	// Allocate host vectors as mapped memory
	err = hipHostAlloc(&h_A, NUM_PKTS * sizeof(int), hipHostMallocMapped);
	err = hipHostAlloc(&h_B, NUM_PKTS * sizeof(int), hipHostMallocMapped);

	// Allocate the host and device flags (host memory versions)
	err = hipHostAlloc(&h_flag_1, sizeof(int), hipHostMallocMapped);
	err = hipHostAlloc(&h_flag_2, sizeof(int), hipHostMallocMapped);
		
	CPE(err != hipSuccess, "Could not allocate managed memory\n", -1);

	assert(h_A != NULL);
	assert(h_B != NULL);
	assert(h_flag_1 != NULL);
	assert(h_flag_2 != NULL);

	// Zero out the mapped memory vectors
	h_flag_1[0] = 0;
	h_flag_2[0] = 0;
	for(int j = 0; j < NUM_PKTS; j++)	{
		h_A[j] = 0;
		h_B[j] = 0;
	}

	// Get device pointer for mapped memory
	err = hipHostGetDevicePointer((void **) &d_A, (void *) h_A, 0);
	err = hipHostGetDevicePointer((void **) &d_B, (void *) h_B, 0);
	err = hipHostGetDevicePointer((void **) &d_flag_1, (void *) h_flag_1, 0);
	err = hipHostGetDevicePointer((void **) &d_flag_2, (void *) h_flag_2, 0);

	CPE(err != hipSuccess, "Could not get device pointer for mapped memory\n", -1);

	// Launch the CPU code
	pthread_create(&thread, NULL, gpu_run, NULL);

	// Launch the kernel once
	printf("Launching CUDA kernel\n");
	int threadsPerBlock = NUM_PKTS;
	int blocksPerGrid = (NUM_PKTS + threadsPerBlock - 1) / threadsPerBlock;
	hipStream_t my_stream;
	err = hipStreamCreate(&my_stream);
	CPE(err != hipSuccess, "Failed to create cudaStream\n", -1);

	vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, my_stream>>>(d_A, d_B, d_flag_1, d_flag_2, NUM_PKTS);
	hipStreamQuery(my_stream);

	printf("Waiting for CPU thread to finish\n");
	pthread_join(thread, NULL);

	// Free allocated mapped memory
	hipHostFree((void *) h_A);
	hipHostFree((void *) h_B);

	// Reset the device and exit
	err = hipDeviceReset();
	CPE(err != hipSuccess, "Failed to de-initialize the device\n", -1);

	printf("Done\n");
	return 0;
}


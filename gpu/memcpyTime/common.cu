#include "common.h"

void printDeviceProperties()
{
	struct hipDeviceProp_t deviceProp;
	int ret = hipGetDeviceProperties(&deviceProp, 0);
	CPE(ret != hipSuccess, "Get Device Properties failed\n", -1);

	printf("Device name: %s\n", deviceProp.name);
	printf("Total global memory: %lu bytes\n", deviceProp.totalGlobalMem);
	printf("Warp size: %d\n", deviceProp.warpSize);
	printf("Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);

	printf("Multi-processor count: %d\n", deviceProp.multiProcessorCount);
	printf("Threads per multi-processor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
}

long long get_cycles()
{
	unsigned low, high;
	unsigned long long val;
	asm volatile ("rdtsc" : "=a" (low), "=d" (high));
	val = high;
	val = (val << 32) | low;
	return val;
}

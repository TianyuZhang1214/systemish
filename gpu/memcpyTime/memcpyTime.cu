#include "common.h"

void gpu_run(int *h_A)
{
	int *d_A = NULL;
	int err = hipSuccess;
	long long startCycles = 0, endCycles = 0, totCycles = 0;
	int i = 0;

	err = hipMalloc((void **) &d_A, NUM_PKTS * sizeof(int));
	CPE(err != hipSuccess, "Failed to hipMalloc\n", -1);

	for(i = 0; i < ITERS; i ++) {
		startCycles = get_cycles();
		err = hipMemcpy(d_A, h_A, NUM_PKTS * sizeof(int), hipMemcpyHostToDevice);
		endCycles = get_cycles();
		totCycles += (endCycles - startCycles);
	}

	printf("Averages: cycles = %lld, nanoseconds = %f ns\n", totCycles / ITERS,
		totCycles / (ITERS * 2.7));

	CPE(err != hipSuccess, "Failed to copy to device memory\n", -1);

	err = hipFree(d_A);
	CPE(err != hipSuccess, "Failed to hipFree\n", -1);
}

int main(void)
{
	int err = hipSuccess;

	printDeviceProperties();

	// Allocate host vectors
	int *h_A = (int *) malloc(NUM_PKTS * sizeof(int));

	// Verify that allocations succeeded
	if (h_A == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors
	for (int i = 0; i < NUM_PKTS; ++i)	{
		h_A[i] = i;
	}

	gpu_run(h_A);
	
	// Free host memory
	free(h_A);

	// Reset the device and exit
	err = hipDeviceReset();
	CPE(err != hipSuccess, "Failed to de-initialize the device\n", -1);

	printf("Done\n");
	return 0;
}


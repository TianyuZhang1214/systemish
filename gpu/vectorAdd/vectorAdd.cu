#include "hip/hip_runtime.h"
#include "common.h"

__global__ void
vectorAdd(const float *A, const float *B, float *C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}

int main(void)
{
	int err, N = 500000;
	struct timespec start, end;

	printDeviceProperties();

	printf("[Vector addition of %d elements]\n", N);

	// Allocate host vectors
	float *h_A = (float *) malloc(N * sizeof(float));
	float *h_B = (float *) malloc(N * sizeof(float));
	float *h_C = (float *) malloc(N * sizeof(float));

	// Verify that allocations succeeded
	if (h_A == NULL || h_B == NULL || h_C == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors
	for (int i = 0; i < N; ++i)	{
		h_A[i] = rand() / (float) RAND_MAX;
		h_B[i] = rand() / (float) RAND_MAX;
	}

	// Start the clock
	clock_gettime(CLOCK_REALTIME, &start);

	// Allocate the device input vector A
	float *d_A = NULL, *d_B = NULL, *d_C = NULL;

	err = hipMalloc((void **)&d_A, N * sizeof(float));
	CPE(err != hipSuccess, "Failed to allocate d_A\n", -1);

	err = hipMalloc((void **)&d_B, N * sizeof(float));
	CPE(err != hipSuccess, "Failed to allocate d_B\n", -1);

	err = hipMalloc((void **)&d_C, N * sizeof(float));
	CPE(err != hipSuccess, "Failed to allocate d_C\n", -1);

	// Copy A and B to device memory
	printf("Copy input data from the host memory to the CUDA device\n");

	err = hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
	CPE(err != hipSuccess, "Failed to copy d_A to device memory\n", -1);

	err = hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);
	CPE(err != hipSuccess, "Failed to copy d_B to device memory\n", -1);

	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

	vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	clock_gettime(CLOCK_REALTIME, &end);

	err = hipGetLastError();
	CPE(err != hipSuccess, "Failed to launch vectorAdd kernel\n", -1);

	// Copy back the result
	printf("Copy output data from the CUDA device to the host memory\n");
	err = hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);
	CPE(err != hipSuccess, "Failed to copy C from device to host\n", -1);

	double time = (double) (end.tv_nsec - start.tv_nsec) / 1000000000 + 
		(end.tv_sec - start.tv_sec);
	printf("Time = %f\n", time);

	// Verify that the result vector is correct
	for (int i = 0; i < N; ++i) {
		if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
	}

	printf("Test PASSED\n");

	// Free device global memory
	err = hipFree(d_A);
	CPE(err != hipSuccess, "Failed to free d_A\n", -1);

	err = hipFree(d_B);
	CPE(err != hipSuccess, "Failed to free d_B\n", -1);

	err = hipFree(d_C);
	CPE(err != hipSuccess, "Failed to free d_C\n", -1);

	// Free host memory
	free(h_A);
	free(h_B);
	free(h_C);

	// Reset the device and exit
	err = hipDeviceReset();
	CPE(err != hipSuccess, "Failed to de-initialize the device\n", -1);

	printf("Done\n");
	return 0;
}


#include "common.h"

void printDeviceProperties()
{
	struct hipDeviceProp_t deviceProp;
	int ret = hipGetDeviceProperties(&deviceProp, 0);
	CPE(ret != hipSuccess, "Get Device Properties failed\n", -1);

	printf("Device name: %s\n", deviceProp.name);
	printf("Total global memory: %lu bytes\n", deviceProp.totalGlobalMem);
	printf("Warp size: %d\n", deviceProp.warpSize);
	printf("Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
}

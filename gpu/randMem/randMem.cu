#include "hip/hip_runtime.h"
#include "common.h"

__global__ void
vectorAdd(int *pkts, const int *log)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < NUM_PKTS) {
		int j;
		for(j = 0; j < DEPTH; j ++) {
			pkts[i] = log[pkts[i]];
		}
	}
}

void cpu_run(int *pkts, int *log)
{
	int i;
	struct timespec start, end;
	clock_gettime(CLOCK_REALTIME, &start);

	for(i = 0; i < NUM_PKTS; i ++) {
		int j;
		for(j = 0; j < DEPTH; j ++) {
			pkts[i] = log[pkts[i]];
		}
	}

	clock_gettime(CLOCK_REALTIME, &end);
	double time = (double) (end.tv_nsec - start.tv_nsec) / 1000000000 + 
		(end.tv_sec - start.tv_sec);
	printf("CPU time = %f\n", time);
}

void gpu_run(int *h_pkts, int *h_log)
{
	struct timespec start, end;
	int *d_pkts = NULL, *d_log = NULL;
	int err = hipSuccess;

	err = hipMalloc((void **) &d_pkts, NUM_PKTS * sizeof(int));
	err = hipMalloc((void **) &d_log, LOG_CAP * sizeof(int));
	CPE(err != hipSuccess, "Failed to hipMalloc\n", -1);

	err = hipMemcpy(d_pkts, h_pkts, NUM_PKTS * sizeof(int), hipMemcpyHostToDevice);
	err = hipMemcpy(d_log, h_log, LOG_CAP * sizeof(int), hipMemcpyHostToDevice);
	CPE(err != hipSuccess, "Failed to copy to device memory\n", -1);

	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (NUM_PKTS + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

	// Start the clock
	clock_gettime(CLOCK_REALTIME, &start);

	vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_pkts, d_log);
	hipDeviceSynchronize();

	clock_gettime(CLOCK_REALTIME, &end);
	double time = (double) (end.tv_nsec - start.tv_nsec) / 1000000000 + 
		(end.tv_sec - start.tv_sec);
	printf("GPU time = %f\n", time);

	err = hipGetLastError();
	CPE(err != hipSuccess, "Failed to launch vectorAdd kernel\n", -1);

	// Copy back the result
	printf("Copy output data from the CUDA device to the host memory\n");
	err = hipMemcpy(h_pkts, d_pkts, NUM_PKTS * sizeof(int), hipMemcpyDeviceToHost);
	CPE(err != hipSuccess, "Failed to copy C from device to host\n", -1);

	// Free device global memory
	err = hipFree(d_pkts);
	err = hipFree(d_log);
	CPE(err != hipSuccess, "Failed to hipFree\n", -1);
}

int main(void)
{
	int err = hipSuccess, i;

	printDeviceProperties();

	int *h_pkts_cpu = (int *) malloc(NUM_PKTS * sizeof(int));
	int *h_pkts_gpu = (int *) malloc(NUM_PKTS * sizeof(int));
	int *h_log = (int *) malloc(LOG_CAP * sizeof(int));

	// Verify that allocations succeeded
	if (h_pkts_cpu == NULL || h_pkts_gpu == NULL || h_log == NULL) {
		fprintf(stderr, "Failed to allocate host mem!\n");
		exit(-1);
	}
	
	// Initialize packets
	for(i = 0; i < NUM_PKTS; i ++) {
		h_pkts_cpu[i] = rand() % LOG_CAP;
		h_pkts_gpu[i] = h_pkts_cpu[i];
	}

	// Initialize log
	for(i = 0; i < LOG_CAP; i ++) {
		h_log[i] = rand() % LOG_CAP;
	}

	cpu_run(h_pkts_cpu, h_log);
	gpu_run(h_pkts_gpu, h_log);

	// Verify that the result vector is correct
	for(int i = 0; i < NUM_PKTS; i ++) {
		if (h_pkts_cpu[i] != h_pkts_gpu[i]) {
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			fprintf(stderr, "CPU %d, GPU %d\n", h_pkts_cpu[i], h_pkts_gpu[i]);
			exit(-1);
		}
	}

	printf("Test PASSED\n");

	// Free host memory
	free(h_pkts_cpu);
	free(h_pkts_gpu);
	free(h_log);

	// Reset the device and exit
	err = hipDeviceReset();
	CPE(err != hipSuccess, "Failed to de-initialize the device\n", -1);

	printf("Done\n");
	return 0;
}

